#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define MAX_BINS 4096
#define DATA_SIZE 256
#define WARP_SIZE 32
#define MAX_BLOCKS 1024

__global__ static void histogramKernel(unsigned int *inputArray, unsigned int *histogram, int unsigned dataSize, int unsigned binSize)
{
    int th = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int local_histogram[];

    for (int bin = threadIdx.x; bin < binSize; bin += blockDim.x)
    {
        local_histogram[bin] = 0;
    }
    __syncthreads();

    for (int i = th; i < dataSize; i += blockDim.x * gridDim.x)
    {
        atomicAdd(&local_histogram[inputArray[i]], 1);
    }

    __syncthreads();

    for (int bin = th; bin < binSize; bin += blockDim.x)
    {
        atomicAdd(&histogram[bin], local_histogram[bin]);
    }
}

void printResult(unsigned int *result, unsigned int resultSize)
{
    printf("Result : [");
    for (int i = 0; i < resultSize; i++)
    {
        printf("%d", result[i]);
        if (i != resultSize - 1)
        {
            printf("-");
        }
        if (i == resultSize - 1)
        {
            printf("]\n");
        }
    }
}

void printData(unsigned int *data, unsigned int dataSize)
{
    printf("Data to be process : [");
    for (int i = 0; i < dataSize; i++)
    {
        printf("%d", data[i]);
        if (i != dataSize - 1)
        {
            printf("-");
        }
        if (i == dataSize - 1)
        {
            printf("]\n");
        }
    }
}


void histogramWrapper(unsigned int dataSize, unsigned int binSize, int display)
{
    unsigned int *histogram = NULL;
    unsigned int *d_histogram = NULL;
    unsigned int *data = NULL;
    unsigned int *d_data = NULL;
    int threadCount = (dataSize/WARP_SIZE);
    hipEvent_t start_t;
    hipEvent_t start_one;
    hipEvent_t stop_t;
    hipEvent_t stop_one;
 
    // Generate the structures
    data = (unsigned int *)malloc(dataSize * sizeof(unsigned int));
    histogram = (unsigned int *)malloc(binSize * sizeof(unsigned int));
    printf("Generating data...\n");
    // Generate the data
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = rand() % binSize;
    }
    printf("Generation done\n");
    // Print the input
    if (display == 1)
    {
	printData(data, dataSize);
    }

    // Assing memory on device
    checkCudaErrors(hipMalloc((void **)&d_histogram, sizeof(unsigned int) * binSize));
    checkCudaErrors(hipMalloc((void **)&d_data, sizeof(unsigned int) * dataSize));

    // Copy the data
    checkCudaErrors(hipMemcpy(d_data, data, sizeof(unsigned int) * dataSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventCreate(&start_t));
    checkCudaErrors(hipEventCreate(&stop_t));
    checkCudaErrors(hipEventRecord(start_t, NULL));

    // Record the start event
    printf("Lauching kernel on %d threads...\n", threadCount);
    // Launch the kernel
    histogramKernel<<<1, threadCount,sizeof(unsigned int) * binSize>>>(d_data, d_histogram, dataSize, binSize);
    hipDeviceSynchronize();

    // Fetch the result
    printf("Kernel ended\n");
    checkCudaErrors(hipMemcpy(histogram, d_histogram, sizeof(unsigned int) * binSize, hipMemcpyDeviceToHost));

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop_t, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop_t));

    checkCudaErrors(hipEventCreate(&start_one));
    checkCudaErrors(hipEventCreate(&stop_one));
    checkCudaErrors(hipEventRecord(start_one, NULL));

    // Record the start event
    printf("Lauching kernel on 1 thread...\n");
    // Launch the kernel
    histogramKernel<<<1, 1,sizeof(unsigned int) * binSize>>>(d_data, d_histogram, dataSize, binSize);
    hipDeviceSynchronize();

    // Fetch the result
    printf("Kernel ended\n");

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop_one, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop_one));


    float msecTotal_t = 0.0f;
    float msecTotal_one = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal_t, start_t, stop_t));
    checkCudaErrors(hipEventElapsedTime(&msecTotal_one, start_one, stop_one));
    double gigaFlops_t = (dataSize * 1.0e-9f) / (msecTotal_t / 1000.0f);
    double gigaFlops_one = (dataSize * 1.0e-9f) / (msecTotal_one / 1000.0f);

    // Print the output
    if (display == 1)
    {
	printResult(histogram, binSize); 
    }

    // Print time enlapsed
    printf("################\n");
    printf("For %d threads :\nCuda processing time = %.3fms, Performance = %.3f GFLOPS\n",threadCount, msecTotal_t, gigaFlops_t);
    printf("For 1 thread :\nCuda processing time = %.3fms, Performance = %.3f GFLOPS\n", msecTotal_one, gigaFlops_one);
    
    free(histogram);
    free(data);
    hipFree(d_data);
    hipFree(d_histogram);
}

int main(int argc, char **argv)
{
    unsigned int binSize = MAX_BINS;
    unsigned long long u_dataSize = DATA_SIZE;
    int display = 0;
    char *dataSize = NULL;
    
    // Get the device    
    int dev = findCudaDevice(argc, (const char **)argv);
    
    // Get the inputs
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage :\n");
        printf("      -dSize=DATA_SIZE [256] (Length of the vector containing the data)\n");
        printf("      -v (Display the data and the histogram)\n");

        exit(EXIT_SUCCESS);
    }

    printf("CUDA - Histogramming algorithm\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "dSize"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "dSize", &dataSize);
	u_dataSize = atoll(dataSize);

    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        display = 1;
    }


    printf("Length of the data : %lu\n", u_dataSize);
    if (u_dataSize >= 4294967296 || u_dataSize == 0) {
        printf("Error: Data size must be < 4,294,967,296. Actual: %lu\n", u_dataSize);
        exit(EXIT_FAILURE);
    }


    histogramWrapper(u_dataSize, binSize, display);
    
    return EXIT_SUCCESS;
}

