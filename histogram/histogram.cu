#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define MAX_BINS 1024
#define DATA_SIZE 256
#define WARP_SIZE 32
#define MAX_BLOCKS 1024

__global__ 
static void histogramKernel(unsigned int *inputArray, unsigned int *histogram, int unsigned dataSize, int unsigned binSize)
{
    int th = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int local_histogram[];

    for (int bin = threadIdx.x; bin < binSize; bin += blockDim.x)
    {
        local_histogram[bin] = 0;
    }
    __syncthreads();

    for (int i = th; i < dataSize; i += blockDim.x * gridDim.x)
    {
        atomicAdd(&local_histogram[inputArray[i]], 1);
    }

    __syncthreads();

    for (int bin = th; bin < binSize; bin += blockDim.x)
    {
        atomicAdd(&histogram[bin], local_histogram[bin]);
    }
}

__global__ 
static void cleanHistogram(unsigned int *histogram, int unsigned binSize)
{
    for (int bin = threadIdx.x; bin < binSize; bin += blockDim.x)
    {
        histogram[bin] = 0;
    }
    __syncthreads();

}




void printResult(unsigned int *result, unsigned int resultSize, int threadCount)
{
    printf("Result for %d threads: [", threadCount);
    for (int i = 0; i < resultSize; i++)
    {
        printf("%d", result[i]);
        if (i != resultSize - 1)
        {
            printf("-");
        }
        if (i == resultSize - 1)
        {
            printf("]\n");
        }
    }
}

void printData(unsigned int *data, unsigned int dataSize)
{
    printf("Data to be process : [");
    for (int i = 0; i < dataSize; i++)
    {
        printf("%d", data[i]);
        if (i != dataSize - 1)
        {
            printf("-");
        }
        if (i == dataSize - 1)
        {
            printf("]\n");
        }
    }
}

bool compareResults(unsigned int *array1, unsigned int *array2, int size)
{
    for(int i = 0; i<size; i++)
    {
	if (array1[i] != array2[i])
	{
	    return false;
	}
    }
    return true;
}


void histogramWrapper(unsigned int dataSize, unsigned int binSize, int display)
{
    unsigned int *histogram_t = NULL;
    unsigned int *histogram_one = NULL;
    unsigned int *d_histogram = NULL;
    unsigned int *data = NULL;
    unsigned int *d_data = NULL;
    int threadCount = (dataSize/WARP_SIZE);
    hipEvent_t start_t;
    hipEvent_t start_one;
    hipEvent_t stop_t;
    hipEvent_t stop_one;

    // Generate the structures
    data = (unsigned int *)malloc(dataSize * sizeof(unsigned int));
    histogram_t = (unsigned int *)malloc(binSize * sizeof(unsigned int));
    histogram_one = (unsigned int *)malloc(binSize * sizeof(unsigned int));

    if (threadCount > 1024)
    { 
 	threadCount = 1024;
    }

    // Generate the data    
    printf("Generating data...\n");
    srand(time(NULL));
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = rand() % binSize;
    }
    printf("Generation done\n");
    
    // Print the input
    if (display == 1)
    {
	printData(data, dataSize);
    }

    // Assing memory on device
    checkCudaErrors(hipMalloc((void **)&d_histogram, sizeof(unsigned int) * binSize));
    checkCudaErrors(hipMalloc((void **)&d_data, sizeof(unsigned int) * dataSize));

    // Copy the data
    checkCudaErrors(hipMemcpy(d_data, data, sizeof(unsigned int) * dataSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventCreate(&start_t));
    checkCudaErrors(hipEventCreate(&stop_t));
    checkCudaErrors(hipEventRecord(start_t, NULL));

    // Record the start event
    printf("Lauching kernel on %d threads...\n", threadCount);
    // Launch the kernel
    histogramKernel<<<1, threadCount,sizeof(unsigned int) * binSize>>>(d_data, d_histogram, dataSize, binSize);
    printf("%d\n", sizeof(unsigned int) * binSize);
    hipDeviceSynchronize();

    // Fetch the result
    printf("Kernel ended\n");
    checkCudaErrors(hipMemcpy(histogram_t, d_histogram, sizeof(unsigned int) * binSize, hipMemcpyDeviceToHost));

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop_t, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop_t));

    checkCudaErrors(hipEventCreate(&start_one));
    checkCudaErrors(hipEventCreate(&stop_one));
    checkCudaErrors(hipEventRecord(start_one, NULL));

    // Clean d_histogram
    printf("Cleaning GPU's histogram...\n");
    cleanHistogram<<<1, threadCount>>>(d_histogram, binSize);
    hipDeviceSynchronize();
    printf("Cleaning done\n");


    // Record the start event
    printf("Lauching kernel on 1 thread...\n");
    // Launch the kernel
    histogramKernel<<<1, 1,sizeof(unsigned int) * binSize>>>(d_data, d_histogram, dataSize, binSize);
    hipDeviceSynchronize();

    // Fetch the result
    printf("Kernel ended\n");
    checkCudaErrors(hipMemcpy(histogram_one, d_histogram, sizeof(unsigned int) * binSize, hipMemcpyDeviceToHost));

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop_one, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop_one));

    float msecTotal_t = 0.0f;
    float msecTotal_one = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal_t, start_t, stop_t));
    checkCudaErrors(hipEventElapsedTime(&msecTotal_one, start_one, stop_one));
    double gigaFlops_t = (dataSize * 1.0e-9f) / (msecTotal_t / 1000.0f);
    double gigaFlops_one = (dataSize * 1.0e-9f) / (msecTotal_one / 1000.0f);

    // Print the output
    if (display == 1)
    {
	printResult(histogram_t, binSize, threadCount); 
	printResult(histogram_one, binSize, 1); 
    }
    // Compare the results
    printf("################\n");
    if (compareResults(histogram_t, histogram_one, binSize))
    {
	printf("OK : Both histogram match\n");
    }
    else
    {
	printf("NOK : Both histogram don't match\n");
    }
    // Print time enlapsed
    printf("################\n");
    printf("For %d threads :\nCuda processing time = %.3fms, Performance = %.3f GFLOPS\n",threadCount, msecTotal_t, gigaFlops_t);
    printf("For 1 thread :\nCuda processing time = %.3fms, Performance = %.3f GFLOPS\n", msecTotal_one, gigaFlops_one);
    
    free(histogram_t);
    free(histogram_one);
    free(data);
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_histogram));
}

int main(int argc, char **argv)
{
    unsigned int binSize = MAX_BINS;
    unsigned long long u_dataSize = DATA_SIZE;
    int display = 0;
    char *dataSize = NULL;
    hipDeviceProp_t prop;

    system("clear");
    
    // Get the device    
    int dev = findCudaDevice(argc, (const char **)argv);
    hipGetDeviceProperties(&prop, dev);
    printf("%d\n", prop.sharedMemPerMultiprocessor);

    // Get the inputs
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage :\n");
        printf("      -dSize=DATA_SIZE [256] (Length of the vector containing the data)\n");
        printf("      -v (Display the data and the histogram)\n");

        exit(EXIT_SUCCESS);
    }
    printf("CUDA - Histogramming algorithm\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "dSize"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "dSize", &dataSize);
	u_dataSize = atoll(dataSize);

    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        display = 1;
    }


    printf("Length of the data : %lu\n", u_dataSize);
    if (u_dataSize >= 4294967296 || u_dataSize == 0) {
        printf("Error: Data size must be < 4,294,967,296. Actual: %lu\n", u_dataSize);
        exit(EXIT_FAILURE);
    }


    histogramWrapper(u_dataSize, binSize, display);
    
    return EXIT_SUCCESS;
}

