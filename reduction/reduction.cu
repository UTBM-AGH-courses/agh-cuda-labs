#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define MAX_BINS 256

int singleThreadedSum (float tab[], int len)
{
    int res = 0;
    for (int i = 0 ; i < len; i++)
    {
        res += tab[i];
    }
    return res;
}



__global__ 
static void reductionKernel(const float *input, float *output)
{
    extern __shared__ float partSum[];
    unsigned int th = threadIdx.x;
    partSum[th] = input[th];
    partSum[th + blockDim.x] = input[th + blockDim.x];
    for (int stride = blockDim.x; stride > 0 ; stride /= 2)
    {
        __syncthreads();
        partSum[th] += partSum[th+stride];
    }
    __syncthreads();
    if (th == 0){
        output[0] = partSum[0];
    }
}

void reductionWrapper(int dataSize, int display, int threadCount, int blockCount)
{
    float *input = NULL;
    float *dinput = NULL;
    float *doutput = NULL;
    float *output = NULL;
    hipEvent_t start;
    hipEvent_t stop;

    // Generate the structures
    output = (float *)malloc(threadCount * 2 * sizeof(float));
    input = (float *)malloc(threadCount * 2 * sizeof(float));

    // Generate data
    for (int i = 0; i < threadCount * 2; i++)
    {
        input[i] = rand() % MAX_BINS;
    }

    // Assing memory on device
    checkCudaErrors(hipMalloc((void **)&dinput, sizeof(float) * threadCount * 2));
    checkCudaErrors(hipMalloc((void **)&doutput, sizeof(float) * threadCount * 2));
    checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * threadCount * 2, hipMemcpyHostToDevice));


    // Allocating CUDA events that we'll use for timing
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, NULL));

    // Launch the kernel
    reductionKernel<<<blockCount, threadCount, sizeof(float) *  threadCount * 2>>>(dinput, doutput);
    hipDeviceSynchronize();

    // Record stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    // Fetch the data
    checkCudaErrors(hipMemcpy(output, doutput, sizeof(float) * threadCount * 2, hipMemcpyDeviceToHost));

    // Compute results
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    printf("The result of the multithreaded function is: %f \n", output[0]);
    printf("Elapsed Time for reduction function to complete is : %f msec \n", msecTotal);

    // Run on single thread
    int singleThreadRes = singleThreadedSum(input, dataSize);

    printf("The result on the single thread function is: %d \n", singleThreadRes);

    free(input);
    free(output);
    checkCudaErrors(hipFree(dinput));
    checkCudaErrors(hipFree(doutput));
}


int main(int argc, char **argv)
{
    int display = 0;
    int smCount;
    int sharedMemoryPerSm;
    int warpSize;
    int dataSize = 256;
    hipDeviceProp_t prop;

    system("clear");
    
    // Get the device    
    int dev = findCudaDevice(argc, (const char **)argv);
    hipGetDeviceProperties(&prop, dev);
    sharedMemoryPerSm = prop.sharedMemPerMultiprocessor;
    smCount = prop.multiProcessorCount;
    warpSize = prop.warpSize;

    // Get the inputs
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage :\n");
        printf("      -dSize=DATA_SIZE [256] (Length of the vector containing the data)\n");
        printf("      -v (Display the data)\n");

        exit(EXIT_SUCCESS);
    }
    printf("CUDA - Sum reduction algorithm\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "dSize")) 
    {
        dataSize = getCmdLineArgumentInt(argc, (const char **)argv, "dSize");
        if (dataSize > 2048)
        {
            printf("LengthTab is > to the possible number of threads \n");  
            exit(EXIT_FAILURE);
        }
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        display = 1;
    }

    int threadCount = dataSize/2;
    int blockCount = 1;
    reductionWrapper(dataSize, display, threadCount, blockCount);
    
    return EXIT_SUCCESS;
}