#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <numeric> 

#define MAX_BINS 4096


hipError_t customCudaError(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void printData(unsigned int *data, unsigned int dataSize)
{
    printf("Data to be process : [");
    for (int i = 0; i < dataSize; i++)
    {
        printf("%d", data[i]);
        if (i != dataSize - 1)
        {
            printf("-");
        }
        if (i == dataSize - 1)
        {
            printf("]\n");
        }
    }
}

__global__
void reductionKernel(unsigned int *data, unsigned int dataSize, unsigned int* globalData)
{

    extern __shared__ unsigned int local_sum[];
    unsigned int th = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Fill with 0 the outbound values 
    local_sum[th] = (i < dataSize ? data[i] : 0);
    local_sum[th + blockDim.x] = (i + blockDim.x*gridDim.x < dataSize ? data[i + blockDim.x* gridDim.x] : 0);

    __syncthreads();

    // Reduction Loop , Interleaved Addressing
    for (unsigned int stride = 1; stride < blockDim.x*2; stride *= 2)
    {
        int index = 2 * stride * th;

        if (index < blockDim.x*2)
            local_sum[index] += local_sum[index + stride];

        __syncthreads();
    }

    // Commit to vram 
    if (th == 0)
    {
        globalData[blockIdx.x] = local_sum[0];
    }
}


unsigned int* reductionWrapper(unsigned int* data, unsigned int dataSize, int threadCount, int blockCount)
{
    unsigned int* finalSum = NULL;
    unsigned int* d_finalSum;
    unsigned int* d_data;
    hipEvent_t start;
    hipEvent_t stop;

    // Create structures
    finalSum = (unsigned int *)malloc(sizeof(unsigned int)*blockCount);

    // Assign data into the device
    customCudaError(hipMalloc((void**)&d_finalSum, blockCount*sizeof(unsigned int)));
    customCudaError(hipMalloc((void**)&d_data, dataSize*sizeof(unsigned int)));

    // Copy the data
    customCudaError(hipMemcpy(d_data, data, sizeof(unsigned int) * dataSize, hipMemcpyHostToDevice));
        
    // Record the start event for the first kernel
    customCudaError(hipEventCreate(&start));
    customCudaError(hipEventCreate(&stop));
    customCudaError(hipEventRecord(start, NULL));

    // Run the kernel
    printf("Lauching kernel on %d threads / %d blocks...\n", threadCount, blockCount);
    reductionKernel<<<blockCount, threadCount, 2*threadCount*sizeof(unsigned int)>>>(d_data, dataSize, d_finalSum);
    customCudaError(hipDeviceSynchronize());
    printf("Kernel ended\n");

    // Fetch the results
    customCudaError(hipMemcpy(finalSum, d_finalSum, sizeof(unsigned int) * blockCount, hipMemcpyDeviceToHost));     

    // Record the stop event for the first event
    customCudaError(hipEventRecord(stop, NULL)); 
    customCudaError(hipEventSynchronize(stop));

    printf("################\n");
    float msecTotal = 0.0f;
    customCudaError(hipEventElapsedTime(&msecTotal, start, stop));
    double gigaFlops = (dataSize * 1.0e-9f) / (msecTotal / 1000.0f);
    printf("Cuda processing time = %.3fms, Performance = %.3f GFlop/s\n",threadCount, msecTotal, gigaFlops);

    // Free the memory
    customCudaError(hipFree(d_finalSum));
    customCudaError(hipFree(d_data));

    return finalSum;
}

int main(int argc, char** argv)
{
    unsigned int* data = NULL;
    int smCount;
    int sharedMemoryPerSm;
    int warpSize;
    unsigned int dataSize = 61;
    int display = 0;
    unsigned int hostResult = 0;
    hipDeviceProp_t prop;

    system("clear");

    // Get the device    
    int dev = findCudaDevice(argc, (const char **)argv);
    hipGetDeviceProperties(&prop, dev);
    sharedMemoryPerSm = prop.sharedMemPerMultiprocessor;
    smCount = prop.multiProcessorCount;
    warpSize = prop.warpSize;

    // Get the inputs
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage :\n");
        printf("      -dSize=DATA_SIZE [256] (Length of the vector containing the data < 10^8)\n");
        printf("      -verbose (Display the data and the histogram)\n");

        exit(EXIT_SUCCESS);
    }
    printf("CUDA - Redution algorithm\n");

    // Init Data Size 
    if (checkCmdLineFlag(argc, (const char**)argv, "dSize")) 
    {
        dataSize = getCmdLineArgumentInt(argc, (const char**)argv, "dSize");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        display = 1;
    }

    // Allocating memory space for data
    data = (unsigned int *)malloc(sizeof(unsigned int)*dataSize);

    // Generate the data
    printf("Generating data...\n");
    srand(time(NULL));
    for (int i = 0; i < dataSize; ++i)
    {
        data[i] = rand() % MAX_BINS;
        hostResult += data[i];
    }
    printf("Generation done\n");

    // Print the input
    if (display == 1)
    {
	    printData(data, dataSize);
    }

    unsigned int threadCount = 16; 
    unsigned int blockCount  = 2;
    
    unsigned int* finalSum = reductionWrapper(data, dataSize, threadCount, blockCount);

    for (int i = 0; i < blockCount; ++i)
    {
        printf("%d\n", finalSum[i]);
    }

    unsigned int deviceResult = std::accumulate(data, data + dataSize, (unsigned int)0);

    // Compare the results
    printf("################\n");
    if (hostResult == deviceResult)
    {
        printf("OK : Both histogram match\n");
    }
    else
    {
        printf("NOK : Both histogram don't match\n");
    }

    // Cuda free
    free(data);
    free(finalSum);

    exit(EXIT_SUCCESS);
}